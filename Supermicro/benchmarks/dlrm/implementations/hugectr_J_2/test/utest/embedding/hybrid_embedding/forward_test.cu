#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/utils.cuh"
#include "hybrid_embedding_cpu.hpp"
#include "test_common.cuh"

/****************** Frequent and infrequent forward network ******************/

template <typename dtype, typename emtype>
class ForwardNetworkTest : public HybridEmbeddingUnitTest<dtype, emtype> {
 protected:
  bool single_node;

 public:
  ForwardNetworkTest(const HybridEmbeddingConfig<dtype> config, size_t batch_size, bool single_node,
                     size_t seed = 1234ll)
      : HybridEmbeddingUnitTest<dtype, emtype>(config, batch_size, seed),
        single_node(single_node) {}

  void run() {
    uint32_t local_batch_size = ceildiv<uint32_t>(this->batch_size, this->num_instances);

    /* Compute expected results on host */
    HybridEmbeddingCpu<dtype, emtype> cpu_embedding(this->config, this->batch_size,
                                                    this->category_location,
                                                    this->category_frequent_index, this->samples);
    cpu_embedding.generate_embedding_vectors();
    cpu_embedding.forward_network();
    if (!single_node) {
      cpu_embedding.calculate_infrequent_model_indices();
      cpu_embedding.forward_a2a_messages();
    }

    /* Tensors for the interaction layer input and messages */
    std::shared_ptr<GeneralBuffer2<CudaAllocator>> buff = GeneralBuffer2<CudaAllocator>::create();
    std::vector<Tensor2<emtype>> interaction_layer_input(this->num_instances);
    for (size_t i = 0; i < this->num_instances; i++) {
      buff->reserve({local_batch_size * this->config.num_tables, this->config.embedding_vec_size},
                    &interaction_layer_input[i]);
    }
    std::vector<Tensor2<emtype>> received_messages(this->num_instances);
    for (size_t i = 0; i < this->num_instances; i++) {
      buff->reserve({local_batch_size * this->config.num_tables, this->config.embedding_vec_size},
                    &received_messages[i]);
    }
    buff->allocate();

    /* Frequent and infrequent forward_network */
    this->build_infrequent();
    this->build_frequent();
    std::vector<std::vector<emtype>> h_interaction_layer_input(this->num_instances);
    for (size_t i = 0; i < this->num_instances; i++) {
      upload_tensor(cpu_embedding.frequent_embedding_vectors[i],
                    this->frequent_embeddings[i].frequent_embedding_vectors_, this->stream);
      upload_tensor(cpu_embedding.infrequent_embedding_vectors[i],
                    this->infrequent_embeddings[i].infrequent_embedding_vectors_, this->stream);
    }
    for (size_t i = 0; i < this->num_instances; i++) {
      this->infrequent_embeddings[i].calculate_network_indices(this->stream);
      this->frequent_embeddings[i].calculate_frequent_sample_indices(this->stream);
      this->frequent_embeddings[i].forward_network(interaction_layer_input[i].get_ptr(),
                                                   this->stream);
      if (single_node) {
        this->infrequent_embeddings[i].forward_network_direct(interaction_layer_input[i].get_ptr(),
                                                              this->stream);
      } else {
        upload_tensor(cpu_embedding.forward_received_messages[i], received_messages[i],
                      this->stream);
        this->infrequent_embeddings[i].forward_network(
            received_messages[i].get_ptr(), interaction_layer_input[i].get_ptr(), this->stream);
      }

      download_tensor(h_interaction_layer_input[i], interaction_layer_input[i], this->stream);
    }

    /* Compare */
    for (size_t i = 0; i < this->num_instances; i++) {
      ASSERT_TRUE(compare_array(
          local_batch_size * this->config.num_tables * this->config.embedding_vec_size,
          h_interaction_layer_input[i].data(), cpu_embedding.interaction_layer_input[i].data(),
          1e-2));
    }
  }
};

/************** Frequent embedding forward model (single node) **************/

template <typename dtype, typename emtype>
class FrequentForwardModelTest : public HybridEmbeddingUnitTest<dtype, emtype> {
 protected:
 public:
  FrequentForwardModelTest(const HybridEmbeddingConfig<dtype> config, size_t batch_size,
                           size_t seed = 1234ll)
      : HybridEmbeddingUnitTest<dtype, emtype>(config, batch_size, seed) {}

  void run() {
    uint32_t local_batch_size = ceildiv<uint32_t>(this->batch_size, this->num_instances);

    HybridEmbeddingCpu<dtype, emtype> cpu_embedding(this->config, this->batch_size,
                                                    this->category_location,
                                                    this->category_frequent_index, this->samples);
    cpu_embedding.calculate_frequent_network_cache_indices();
    cpu_embedding.generate_embedding_vectors();
    cpu_embedding.generate_gradients();
    cpu_embedding.frequent_reduce_gradients();

    /* Tensors for the gradients */
    std::shared_ptr<GeneralBuffer2<CudaAllocator>> buff = GeneralBuffer2<CudaAllocator>::create();
    std::vector<Tensor2<emtype>> gradients(this->num_instances);
    for (size_t i = 0; i < this->num_instances; i++) {
      buff->reserve({local_batch_size * this->config.num_tables, this->config.embedding_vec_size},
                    &gradients[i]);
    }
    buff->allocate();

    /* Frequent update_model */
    this->build_frequent();
    for (size_t i = 0; i < this->num_instances; i++) {
      upload_tensor(cpu_embedding.frequent_embedding_vectors[i],
                    this->frequent_embeddings[i].frequent_embedding_vectors_, this->stream);
      upload_tensor(cpu_embedding.gradients[i], gradients[i], this->stream);
    }
    for (size_t i = 0; i < this->num_instances; i++) {
      this->frequent_embeddings[i].calculate_network_cache_mask(this->stream);
      this->frequent_embeddings[i].calculate_network_cache_indices(this->stream);
      this->frequent_embeddings[i].calculate_frequent_sample_indices(this->stream);
      this->frequent_embeddings[i].local_reduce(gradients[i].get_ptr(), this->stream, false);
      this->frequent_embeddings[i].update_model_direct(this->config.lr, this->stream);
    }

    /* Frequent forward_model */
    for (size_t i = 0; i < this->num_instances; i++) {
      this->frequent_embeddings[i].forward_model(this->stream);
    }

    std::vector<std::vector<float>> updated_vectors(this->num_instances);
    for (size_t i = 0; i < this->num_instances; i++) {
      download_tensor(updated_vectors[i], this->frequent_embeddings[i].frequent_embedding_vectors_,
                      this->stream);
    }

    /* Reference update_model */
    cpu_embedding.frequent_update_single_node();

    /* Reference forward_model */
    cpu_embedding.frequent_forward_model();

    /* Compare */
    for (size_t i = 0; i < this->num_instances; i++) {
      updated_vectors[i].resize(this->config.num_frequent * this->config.embedding_vec_size);
      EXPECT_THAT(updated_vectors[i],
                  ::testing::Pointwise(::testing::FloatNear(1e-2),
                                       cpu_embedding.frequent_embedding_vectors[i]));
    }
  }
};

/**************************** Test instantiations ****************************/

static const HybridEmbeddingConfig<uint32_t> config_uint32 = {4, 32, 10, 128, 1000, 128, 0.5f};
static const HybridEmbeddingConfig<long long> config_int64 = {4, 32, 10, 128, 1000, 128, 0.5f};
static const HybridEmbeddingConfig<uint32_t> config_uint32_single_node = {1,    8,   10,  128,
                                                                          1000, 128, 0.5f};
static const HybridEmbeddingConfig<long long> config_int64_single_node = {1,    8,   10,  128,
                                                                          1000, 128, 0.5f};

// Edge cases: no frequent, all frequent
static const HybridEmbeddingConfig<uint32_t> config_no_freq = {4, 32, 10, 128, 1000, 0, 0.5f};
static const HybridEmbeddingConfig<uint32_t> config_all_freq = {4, 32, 10, 128, 1000, 1000, 0.5f};
static const HybridEmbeddingConfig<uint32_t> config_no_freq_single_node = {1,    8, 10,  128,
                                                                           1000, 0, 0.5f};
static const HybridEmbeddingConfig<uint32_t> config_all_freq_single_node = {1,    8,    10,  128,
                                                                            1000, 1000, 0.5f};

/* hybrid_embedding_forward_network_test */

TEST(hybrid_embedding_forward_network_test, uint32_half_64) {
  ForwardNetworkTest<uint32_t, __half>(config_uint32, 64, false).run();
}

TEST(hybrid_embedding_forward_network_test, int64_half_64) {
  ForwardNetworkTest<long long, __half>(config_int64, 64, false).run();
}

TEST(hybrid_embedding_forward_network_test, uint32_half_2048) {
  ForwardNetworkTest<uint32_t, __half>(config_uint32, 2048, false).run();
}

TEST(hybrid_embedding_forward_network_test, int64_half_2048) {
  ForwardNetworkTest<long long, __half>(config_int64, 2048, false).run();
}

TEST(hybrid_embedding_forward_network_test, uint32_float_64) {
  ForwardNetworkTest<uint32_t, float>(config_uint32, 64, false).run();
}

TEST(hybrid_embedding_forward_network_test, int64_float_64) {
  ForwardNetworkTest<long long, float>(config_int64, 64, false).run();
}

TEST(hybrid_embedding_forward_network_test, uint32_float_2048) {
  ForwardNetworkTest<uint32_t, float>(config_uint32, 2048, false).run();
}

TEST(hybrid_embedding_forward_network_test, int64_float_2048) {
  ForwardNetworkTest<long long, float>(config_int64, 2048, false).run();
}

TEST(hybrid_embedding_forward_network_test, uint32_float_128_no_freq) {
  ForwardNetworkTest<uint32_t, float>(config_no_freq, 128, false).run();
}

TEST(hybrid_embedding_forward_network_test, uint32_float_128_all_freq) {
  ForwardNetworkTest<uint32_t, float>(config_all_freq, 128, false).run();
}

/* hybrid_embedding_forward_network_single_node_test */

TEST(hybrid_embedding_forward_network_single_node_test, uint32_half_64) {
  ForwardNetworkTest<uint32_t, __half>(config_uint32_single_node, 64, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, int64_half_64) {
  ForwardNetworkTest<long long, __half>(config_int64_single_node, 64, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, uint32_half_2048) {
  ForwardNetworkTest<uint32_t, __half>(config_uint32_single_node, 2048, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, int64_half_2048) {
  ForwardNetworkTest<long long, __half>(config_int64_single_node, 2048, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, uint32_float_64) {
  ForwardNetworkTest<uint32_t, float>(config_uint32_single_node, 64, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, int64_float_64) {
  ForwardNetworkTest<long long, float>(config_int64_single_node, 64, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, uint32_float_2048) {
  ForwardNetworkTest<uint32_t, float>(config_uint32_single_node, 2048, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, int64_float_2048) {
  ForwardNetworkTest<long long, float>(config_int64_single_node, 2048, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, uint32_float_128_no_freq) {
  ForwardNetworkTest<uint32_t, float>(config_no_freq_single_node, 128, true).run();
}

TEST(hybrid_embedding_forward_network_single_node_test, uint32_float_128_all_freq) {
  ForwardNetworkTest<uint32_t, float>(config_all_freq_single_node, 128, true).run();
}

/* hybrid_embedding_frequent_forward_model_test */

TEST(hybrid_embedding_frequent_forward_model_test, uint32_half_64) {
  FrequentForwardModelTest<uint32_t, __half>(config_uint32_single_node, 64).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, int64_half_64) {
  FrequentForwardModelTest<long long, __half>(config_int64_single_node, 64).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, uint32_half_2048) {
  FrequentForwardModelTest<uint32_t, __half>(config_uint32_single_node, 2048).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, int64_half_2048) {
  FrequentForwardModelTest<long long, __half>(config_int64_single_node, 2048).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, uint32_float_64) {
  FrequentForwardModelTest<uint32_t, float>(config_uint32_single_node, 64).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, int64_float_64) {
  FrequentForwardModelTest<long long, float>(config_int64_single_node, 64).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, uint32_float_2048) {
  FrequentForwardModelTest<uint32_t, float>(config_uint32_single_node, 2048).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, int64_float_2048) {
  FrequentForwardModelTest<long long, float>(config_int64_single_node, 2048).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, uint32_float_128_no_freq) {
  FrequentForwardModelTest<uint32_t, float>(config_no_freq_single_node, 128).run();
}

TEST(hybrid_embedding_frequent_forward_model_test, uint32_float_128_all_freq) {
  FrequentForwardModelTest<uint32_t, float>(config_all_freq_single_node, 128).run();
}
